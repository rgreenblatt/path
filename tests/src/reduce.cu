#include "hip/hip_runtime.h"
#ifndef CPU_ONLY
#include "execution_model/host_device_vector.h"
#include "execution_model/host_vector.h"
#include "kernel/kernel_launch.h"
#include "kernel/kernel_launch_impl_cpu.h"
#include "kernel/kernel_launch_impl_gpu.cuh"
#include "kernel/make_runtime_constants_reduce_launchable.h"
#include "kernel/reduce_samples.cuh"
#include "kernel/runtime_constants_reducer_impl_gpu.cuh"
#include "kernel/work_division.h"
#include "lib/assert.h"
#include "lib/cuda/reduce.cuh"
#include "lib/cuda/utils.h"
#include "lib/span.h"
#include "meta/all_values/tag.h"

#include <gmock/gmock.h>
#include <gtest/gtest.h>

#include <algorithm>
#include <random>

using kernel::WorkDivision;

template <typename T>
__global__ void sum_sub_blocks(Span<const T> in, Span<T> out,
                               unsigned sub_block_size) {
  unsigned thread_idx = threadIdx.x;
  unsigned block_idx = blockIdx.x;
  unsigned block_size = blockDim.x;
  unsigned overall_idx = thread_idx + block_idx * block_size;
  unsigned sub_block_idx = overall_idx / sub_block_size;
  auto add = [](const T &lhs, const T &rhs) { return lhs + rhs; };
  const T total = sub_block_reduce<T>(in[overall_idx], add, thread_idx,
                                      block_size, sub_block_size);
  if (thread_idx % sub_block_size == 0) {
    out[sub_block_idx] = total;
  }
}

#define EXPECT_FLOATS_EQ(expected, actual)                                     \
  EXPECT_EQ(expected.size(), actual.size()) << "Sizes differ.";                \
  for (size_t idx = 0; idx < std::min(expected.size(), actual.size());         \
       ++idx) {                                                                \
    EXPECT_FLOAT_EQ(expected[idx], actual[idx])                                \
        << "at index: " << idx << " and line: " << __LINE__;                   \
  }

TEST(Reduce, sum) {
  auto run_test = [](auto dist, auto check_equality) {
    for (unsigned num_locations : {1, 2, 3, 7, 8, 17, 32, 256}) {
      for (unsigned samples_per : {1, 2, 3, 7, 8, 32, 37, 49, 128, 189, 256})
        for (unsigned block_size : {32, 128, 256, 1024}) {
          for (unsigned base_target_samples_per_thread : {1, 2, 3, 5}) {
            const unsigned size = num_locations * samples_per;

            // avoid this test taking too long
            if (size > 4096) {
              continue;
            }

            const unsigned target_x_block_size = block_size;
            unsigned target_samples_per_thread = base_target_samples_per_thread;
            WorkDivision division;
            do {
              division = WorkDivision({block_size, target_x_block_size, true,
                                       target_samples_per_thread},
                                      samples_per, num_locations, 1);
              target_samples_per_thread *= 2;
            } while (division.num_sample_blocks() != 1);
            ASSERT_EQ(division.num_sample_blocks(), 1);

            std::mt19937 gen(testing::UnitTest::GetInstance()->random_seed());

            using T = std::decay_t<decltype(dist(gen))>;

            HostDeviceVector<T> vals(size);

            std::generate(vals.begin(), vals.end(),
                          [&]() { return dist(gen); });

            HostDeviceVector<T> out_vals(num_locations);

            Span<const T> in = vals;
            Span<T> out = out_vals;

            bool use_direct_approach =
                block_size % samples_per == 0 && size % block_size == 0;
            if (use_direct_approach) {
              unsigned num_blocks = size / block_size;
              always_assert(num_blocks * block_size == size);
              sum_sub_blocks<T>
                  <<<num_blocks, block_size>>>(in, out, samples_per);
            }

            auto division_run = [&](auto tag, Span<T> out_div) {
              constexpr ExecutionModel exec = tag;

              kernel::KernelLaunch<exec>::run(
                  division, 0, division.total_num_blocks(),
                  kernel::make_runtime_constants_reduce_launchable<exec, T>(
                      [=] HOST_DEVICE(const WorkDivision &division,
                                      const kernel::GridLocationInfo &info,
                                      const unsigned /*block_idx*/,
                                      const unsigned /*thread_idx*/,
                                      const auto &, auto &interactor) {
                        auto [start_sample, end_sample, j, unused] = info;

                        T total = 0;
                        for (unsigned i = start_sample; i < end_sample; ++i) {
                          total += in[i + j * samples_per];
                        }

                        auto add = [](const T &lhs, const T &rhs) {
                          return lhs + rhs;
                        };
                        auto op = interactor.reduce(
                            total, add, division.sample_block_size());
                        if (op.has_value()) {
                          out_div[j] = *op;
                        }
                      }));
            };

            HostDeviceVector<T> out_vals_division(num_locations);
            HostVector<T> out_vals_division_cpu(num_locations);

            division_run(tag_v<ExecutionModel::GPU>, out_vals_division);
            division_run(tag_v<ExecutionModel::CPU>, out_vals_division_cpu);

            std::vector<T> expected(num_locations, 0.f);

            for (unsigned location = 0; location < num_locations; ++location) {
              for (unsigned i = location * samples_per;
                   i < (location + 1) * samples_per; ++i) {
                expected[location] += vals[i];
              }
            }

            if (use_direct_approach) {
              check_equality(expected, out_vals);
            }
            check_equality(expected, out_vals_division);
            check_equality(expected, out_vals_division_cpu);
          }
        }
    }
  };

  run_test(std::uniform_real_distribution<double>(0.0, 1.0),
           [](const auto &expected, const auto &actual) {
             EXPECT_FLOATS_EQ(expected, actual);
           });
  run_test(std::uniform_int_distribution<int>(-100, 100),
           [](const auto &expected, const auto &actual) {
             EXPECT_THAT(expected, testing::ElementsAreArray(actual));
           });
}
#endif
