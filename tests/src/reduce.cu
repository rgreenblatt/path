#include "hip/hip_runtime.h"
#ifndef CPU_ONLY
#include "execution_model/host_device_vector.h"
#include "lib/assert.h"
#include "lib/cuda/reduce.cuh"
#include "lib/cuda/utils.h"
#include "lib/span.h"
#include "work_division/kernel_launch.h"
#include "work_division/kernel_launch_impl_gpu.cuh"
#include "work_division/reduce_samples.cuh"
#include "work_division/work_division.h"

#include <gmock/gmock.h>
#include <gtest/gtest.h>

#include <algorithm>
#include <random>

using work_division::WorkDivision;

template <typename T>
__global__ void sum_sub_blocks(Span<const T> in, Span<T> out,
                               unsigned sub_block_size) {
  unsigned thread_idx = threadIdx.x;
  unsigned block_idx = blockIdx.x;
  unsigned block_size = blockDim.x;
  unsigned overall_idx = thread_idx + block_idx * block_size;
  unsigned sub_block_idx = overall_idx / sub_block_size;
  auto add = [](const T &lhs, const T &rhs) { return lhs + rhs; };
  const T total = sub_block_reduce<T>(in[overall_idx], add, thread_idx,
                                      block_size, sub_block_size);
  if (thread_idx % sub_block_size == 0) {
    out[sub_block_idx] = total;
  }
}

#define EXPECT_FLOATS_EQ(expected, actual)                                     \
  EXPECT_EQ(expected.size(), actual.size()) << "Sizes differ.";                \
  for (size_t idx = 0; idx < std::min(expected.size(), actual.size());         \
       ++idx) {                                                                \
    EXPECT_FLOAT_EQ(expected[idx], actual[idx])                                \
        << "at index: " << idx << " and line: " << __LINE__;                   \
  }

TEST(Reduce, sum) {
  auto run_test = [](auto dist, auto check_equality) {
    for (unsigned num_locations : {1, 2, 3, 7, 8, 17, 32, 256}) {
      for (unsigned samples_per : {1, 2, 3, 7, 8, 32, 37, 49, 128, 189, 256})
        for (unsigned block_size : {32, 128, 256, 1024}) {
          for (unsigned base_target_samples_per_thread : {1, 2, 3, 5}) {
            const unsigned size = num_locations * samples_per;

            // avoid this test taking too long
            if (size > 4096) {
              continue;
            }

            const unsigned target_x_block_size = block_size;
            unsigned target_samples_per_thread = base_target_samples_per_thread;
            WorkDivision division;
            do {
              division = WorkDivision({block_size, target_x_block_size, true,
                                       target_samples_per_thread},
                                      samples_per, num_locations, 1);
              target_samples_per_thread *= 2;
            } while (division.num_sample_blocks() != 1);
            ASSERT_EQ(division.num_sample_blocks(), 1);

            std::mt19937 gen(testing::UnitTest::GetInstance()->random_seed());

            using T = std::decay_t<decltype(dist(gen))>;

            HostDeviceVector<T> vals(size);

            std::generate(vals.begin(), vals.end(),
                          [&]() { return dist(gen); });

            HostDeviceVector<T> out_vals(num_locations);

            Span<const T> in = vals;
            Span<T> out = out_vals;

            bool use_direct_approach =
                block_size % samples_per == 0 && size % block_size == 0;
            if (use_direct_approach) {
              unsigned num_blocks = size / block_size;
              always_assert(num_blocks * block_size == size);
              sum_sub_blocks<T>
                  <<<num_blocks, block_size>>>(in, out, samples_per);
            }

            HostDeviceVector<T> out_vals_division(num_locations);

            Span<T> out_division = out_vals_division;

            work_division::KernelLaunch<ExecutionModel::GPU>::run(
                division, 0, division.total_num_blocks(),
                [=] HOST_DEVICE(const WorkDivision &division,
                                const work_division::GridLocationInfo &info,
                                const unsigned /*block_idx*/,
                                const unsigned thread_idx) {
                  auto [start_sample, end_sample, j, unused] = info;

                  T total = 0;
                  for (unsigned i = start_sample; i < end_sample; ++i) {
                    total += in[i + j * samples_per];
                  }

                  auto add = [](const T &lhs, const T &rhs) {
                    return lhs + rhs;
                  };
                  total = reduce_samples(division, total, add, thread_idx);
                  if (division.assign_sample(thread_idx)) {
                    out_division[j] = total;
                  }
                });

            std::vector<T> expected(num_locations, 0.f);

            for (unsigned location = 0; location < num_locations; ++location) {
              for (unsigned i = location * samples_per;
                   i < (location + 1) * samples_per; ++i) {
                expected[location] += vals[i];
              }
            }

            if (use_direct_approach) {
              check_equality(expected, out_vals);
            }
            check_equality(expected, out_vals_division);
          }
        }
    }
  };

  run_test(std::uniform_real_distribution<double>(0.0, 1.0),
           [](const auto &expected, const auto &actual) {
             EXPECT_FLOATS_EQ(expected, actual);
           });
  run_test(std::uniform_int_distribution<int>(-100, 100),
           [](const auto &expected, const auto &actual) {
             EXPECT_THAT(expected, testing::ElementsAreArray(actual));
           });
}
#endif
