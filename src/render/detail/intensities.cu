#include "hip/hip_runtime.h"
#ifndef CPU_ONLY_BUILD
#include "lib/cuda/reduce.h"
#include "render/detail/impl/intensities_impl.h"
#include "render/detail/impl/render_impl.h"

#include <cli/ProgressBar.hpp>

namespace render {
namespace detail {
template <intersect::accel::AccelRef MeshAccel,
          intersect::accel::AccelRef TriAccel, LightSamplerRef L,
          DirSamplerRef D, TermProbRef T, rng::RngRef R>
__global__ void
intensities_global(const ComputationSettings &settings, unsigned start_blocks,
                   const WorkDivision division, unsigned x_dim, unsigned y_dim,
                   unsigned samples_per, const MeshAccel mesh_accel,
                   Span<const TriAccel> tri_accels, const L light_sampler,
                   const D direction_sampler, const T term_prob, const R rng,
                   Span<BGRA> bgras, Span<Eigen::Array3f>,
                   Span<const scene::TriangleData> triangle_data,
                   Span<const material::Material> materials,
                   const Eigen::Affine3f film_to_world) {
  assert(division.num_sample_blocks == 1);

  const unsigned block_idx = blockIdx.x + start_blocks;
  const unsigned thread_idx = threadIdx.x;
  const unsigned block_dim = blockDim.x;

  const unsigned block_idx_sample = block_idx % division.num_sample_blocks;
  const unsigned block_idx_pixel = block_idx / division.num_sample_blocks;
  const unsigned block_idx_x = block_idx_pixel % division.num_x_blocks;
  const unsigned block_idx_y = block_idx_pixel / division.num_x_blocks;

  unsigned work_idx = division.samples_per_thread * thread_idx;

  unsigned sample_block_size = samples_per / division.num_sample_blocks;

  const unsigned work_idx_sample = work_idx % sample_block_size;
  const unsigned work_idx_pixel = work_idx / sample_block_size;
  const unsigned work_idx_x = work_idx_pixel % division.x_block_size;
  const unsigned work_idx_y = work_idx_pixel / division.x_block_size;

  const unsigned start_sample =
      work_idx_sample + block_idx_sample * sample_block_size;
  const unsigned end_sample = start_sample + division.samples_per_thread;
  const unsigned x = work_idx_x + block_idx_x * division.x_block_size;
  const unsigned y = work_idx_y + block_idx_y * division.y_block_size;

  if (x >= x_dim || y >= y_dim) {
    return;
  }

  auto intensity = intensities_impl(
      x, y, start_sample, end_sample, settings, x_dim, y_dim, samples_per,
      mesh_accel, tri_accels, light_sampler, direction_sampler, term_prob, rng,
      triangle_data, materials, film_to_world);

  // below reduction assumes this is the case
  assert(division.num_sample_blocks == 1);

  auto compute_bgras = [&](const auto &reduce_func, unsigned idx) {
    Eigen::Array3f totals;
    for (unsigned axis = 0; axis < 3; axis++) {
      totals[axis] = reduce_func(intensity[axis]);
    }
    if (idx == 0) {
      bgras[x + y * x_dim] = intensity_to_bgr(totals / samples_per);
    }
  };

  auto add = [](auto lhs, auto rhs) { return lhs + rhs; };

  switch (division.sample_reduction_strategy) {
  case ReductionStrategy::Block:
    compute_bgras(
        [&](const float v) {
          return block_reduce(v, add, 0.0f, thread_idx, block_dim);
        },
        thread_idx);
    break;
  case ReductionStrategy::Warp:
    compute_bgras([&](const float v) { return warp_reduce(v, add); },
                  thread_idx % warpSize);
    break;
  case ReductionStrategy::Thread:
    compute_bgras([&](const float v) { return v; }, 0);
    break;
  }
}

template <intersect::accel::AccelRef MeshAccel,
          intersect::accel::AccelRef TriAccel, LightSamplerRef L,
          DirSamplerRef D, TermProbRef T, rng::RngRef R>
void intensities(const ComputationSettings &settings,
                 const WorkDivision &division, unsigned samples_per,
                 unsigned x_dim, unsigned y_dim, const MeshAccel &mesh_accel,
                 Span<const TriAccel> tri_accels, const L &light_sampler,
                 const D &direction_sampler, const T &term_prob, const R &rng,
                 Span<BGRA> pixels, Span<Eigen::Array3f> intensities,
                 Span<const scene::TriangleData> triangle_data,
                 Span<const material::Material> materials,
                 const Eigen::Affine3f &film_to_world) {
  unsigned block_size = division.block_size;

  size_t total_size = size_t(samples_per) * x_dim * y_dim;
  size_t total_items_per_block = division.samples_per_thread * block_size;
  assert(total_size >= total_items_per_block);
  assert(total_size % total_items_per_block == 0);
  size_t total_grid =
      (total_size + total_items_per_block - 1) / total_items_per_block;

  size_t max_launch_size = 2 << 24;

  size_t num_launches = (total_size + max_launch_size - 1) / max_launch_size;
  size_t blocks_per = total_grid / num_launches;

  ProgressBar progress_bar(num_launches, 70);
  progress_bar.display();

  for (unsigned i = 0; i < num_launches; i++) {
    unsigned start = i * blocks_per;
    unsigned end = std::min((i + 1) * blocks_per, total_grid);
    unsigned grid = end - start;
    intensities_global<<<grid, block_size>>>(
        settings, start, division, x_dim, y_dim, samples_per, mesh_accel,
        tri_accels, light_sampler, direction_sampler, term_prob, rng, pixels,
        intensities, triangle_data, materials, film_to_world);

    CUDA_ERROR_CHK(hipDeviceSynchronize());

    ++progress_bar;
    progress_bar.display();
  }

  progress_bar.done();
}

template class RendererImpl<ExecutionModel::GPU>;
} // namespace detail
} // namespace render
#endif
