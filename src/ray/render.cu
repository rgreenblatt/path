#include "lib/unified_memory_vector.h"
#include "ray/intersect.cuh"
#include "ray/lighting.cuh"
#include "ray/render.h"
#include "ray/kdtree.h"

#include <boost/range/adaptor/indexed.hpp>
#include <boost/range/combine.hpp>
#include <thrust/fill.h>

#include <dbg.h>
#include <chrono>

namespace ray {
using namespace detail;

unsigned num_blocks(unsigned size, unsigned block_size) {
  return (size + block_size - 1) / block_size;
};

template <ExecutionModel execution_model>
Renderer<execution_model>::Renderer(unsigned width, unsigned height,
                                    unsigned recursive_iterations,
                                    unsigned x_special, unsigned y_special)
    : width_(width), height_(height), x_special_(x_special),
      y_special_(y_special), recursive_iterations_(recursive_iterations),
      by_type_data_(invoke([&] {
        auto get_by_type = [&](scene::Shape shape_type) {
          return ByTypeData(width, height, shape_type);
        };

        return std::array{
            get_by_type(scene::Shape::Sphere),
            get_by_type(scene::Shape::Cylinder),
            get_by_type(scene::Shape::Cube),
            get_by_type(scene::Shape::Cone),
        };
      })),
      world_space_eyes_(width * height),
      world_space_directions_(width * height), ignores_(width * height),
      color_multipliers_(width * height), disables_(width * height),
      colors_(width * height), bgra_(width * height) {}

template <ExecutionModel execution_model>
template <typename... T>
void Renderer<execution_model>::minimize_intersections(unsigned size,
                                                       T &... values) {
  if constexpr (execution_model == ExecutionModel::GPU) {
    const unsigned minimize_block_size = 256;
    minimize_all_intersections<<<num_blocks(size, minimize_block_size),
                                 minimize_block_size>>>(
        size, values.intersections.data()...);

    CUDA_ERROR_CHK(hipDeviceSynchronize());
  } else {
    minimize_all_intersections_cpu(size, values.intersections.data()...);
  }
}

template <ExecutionModel execution_model>
template <typename... T>
void Renderer<execution_model>::minimize_intersections(ByTypeData &first,
                                                       const T &... rest) {
  minimize_intersections(static_cast<unsigned>(first.intersections.size()),
                         first, rest...);
}

template <ExecutionModel execution_model>
void Renderer<execution_model>::render(
    const scene::Scene &scene, BGRA *pixels,
    const scene::Transform &m_film_to_world) {
  const auto lights = scene.get_lights();
  const unsigned num_lights = scene.get_num_lights();
  const unsigned num_pixels = width_ * height_;

  /* light_shadowed_.resize(num_lights * num_pixels); */

  // TODO
  /* unsigned width_block_size = 4; */
  /* unsigned height_block_size = 4; */
  /* unsigned shapes_block_size = 64; */

  const unsigned width_block_size = 32;
  const unsigned height_block_size = 32;

  const dim3 grid(num_blocks(width_, width_block_size),
                  num_blocks(height_, height_block_size), 1);
  const dim3 block(width_block_size, height_block_size, 1);

  const Eigen::Vector3f world_space_eye = m_film_to_world.translation();

  const scene::Color initial_multiplier = scene::Color::Ones();

  const scene::Color initial_color = scene::Color::Zero();

  // could be made async until...
  if constexpr (execution_model == ExecutionModel::GPU) {
    const unsigned fill_block_size = 256;
    fill<<<num_blocks(num_pixels, fill_block_size), fill_block_size>>>(
        world_space_eyes_.data(), num_pixels, world_space_eye);
    fill<<<num_blocks(num_pixels, fill_block_size), fill_block_size>>>(
        color_multipliers_.data(), num_pixels, initial_multiplier);
    fill<<<num_blocks(num_pixels, fill_block_size), fill_block_size>>>(
        colors_.data(), num_pixels, initial_color);

    initial_world_space_directions<<<grid, block>>>(
        width_, height_, world_space_eye, m_film_to_world,
        world_space_directions_.data());

    CUDA_ERROR_CHK(hipDeviceSynchronize());
  } else {
    std::fill(world_space_eyes_.begin(), world_space_eyes_.end(),
              world_space_eye);
    std::fill(color_multipliers_.begin(), color_multipliers_.end(),
              initial_multiplier);
    std::fill(colors_.begin(), colors_.end(), initial_color);

    initial_world_space_directions_cpu(width_, height_, world_space_eye,
                                       m_film_to_world,
                                       world_space_directions_.data());
  }
    
  auto start_shape = scene.get_shapes();
  const unsigned num_shapes = scene.num_shapes();
  DataType<scene::ShapeData> shapes(num_shapes);

  std::copy(start_shape, start_shape + num_shapes, shapes.begin());

/* #pragma omp parallel for */
  for (auto &data : by_type_data_) {
    const unsigned num_shape = scene.get_num_shape(data.shape_type);
    const unsigned start_shape = scene.get_start_shape(data.shape_type);

    auto kdtree =
        construct_kd_tree(shapes.data() + start_shape, num_shape);
    data.nodes.resize(kdtree.size());
    std::copy(kdtree.begin(), kdtree.end(), data.nodes.begin());
  }

  for (unsigned depth = 0; depth < recursive_iterations_; depth++) {
    bool is_first = depth == 0;

    const auto start_intersect = std::chrono::high_resolution_clock::now();
/* #pragma omp parallel for */
    for (auto &data : by_type_data_) {
      const unsigned start_shape = scene.get_start_shape(data.shape_type);
      if constexpr (execution_model == ExecutionModel::GPU) {
        solve_intersections<<<grid, block>>>(
            width_, height_, start_shape, shapes.data(),
            world_space_eyes_.data(), world_space_directions_.data(),
            data.nodes.data(), data.nodes.size(), ignores_.data(),
            disables_.data(), data.intersections.data(), data.shape_type,
            is_first);
      } else {
        solve_intersections_cpu(
            width_, height_, start_shape, shapes.data(),
            world_space_eyes_.data(), world_space_directions_.data(),
            data.nodes.data(), data.nodes.size(), ignores_.data(),
            disables_.data(), data.intersections.data(), data.shape_type,
            is_first);
      }
    }

    CUDA_ERROR_CHK(hipDeviceSynchronize());
    
    dbg(std::chrono::duration_cast<std::chrono::duration<double>>(
        std::chrono::high_resolution_clock::now() - start_intersect).count());

    // fuse kernel???
    minimize_intersections(by_type_data_[0], by_type_data_[1], by_type_data_[2],
                           by_type_data_[3]);

    auto &best_intersections = by_type_data_[0].intersections;
    
#if 0
    for (auto &data : by_type_data_) {
      const unsigned num_shape = scene.get_num_shape(data.shape_type);
      const unsigned start_shape = scene.get_start_shape(data.shape_type);

      if constexpr (execution_model == ExecutionModel::GPU) {
        check_intersections<<<grid, block>>>(
            width_, height_, num_shape, start_shape, shapes,
            world_space_eyes_.data(), world_space_directions_.data(),
            ignores_.data(), disables_.data(), data.intersections.data(),
            data.shape_type, is_first);
      } else {
        solve_intersections_cpu(width_, height_, num_shape, start_shape, shapes,
                                world_space_eyes_.data(),
                                world_space_directions_.data(), ignores_.data(),
                                disables_.data(), data.intersections.data(),
                                data.shape_type, is_first);
      }
    }
#endif

    const auto start_color = std::chrono::high_resolution_clock::now();
    if constexpr (execution_model == ExecutionModel::GPU) {
      // TODO block etc...
      compute_colors<<<grid, block>>>(
          width_, height_, world_space_eyes_.data(),
          world_space_directions_.data(), ignores_.data(),
          color_multipliers_.data(), disables_.data(),
          best_intersections.data(), shapes.data(), lights, num_lights,
          colors_.data(), is_first, x_special_, y_special_);

      CUDA_ERROR_CHK(hipDeviceSynchronize());
    } else {
      compute_colors_cpu(width_, height_, world_space_eyes_.data(),
                         world_space_directions_.data(), ignores_.data(),
                         color_multipliers_.data(), disables_.data(),
                         best_intersections.data(), shapes.data(), lights,
                         num_lights, colors_.data(), is_first, x_special_,
                         y_special_);
    }
    dbg(std::chrono::duration_cast<std::chrono::duration<double>>(
            std::chrono::high_resolution_clock::now() - start_color)
            .count());
  }

  if constexpr (execution_model == ExecutionModel::GPU) {
    floats_to_bgras<<<grid, block>>>(width_, height_, colors_.data(),
                                     bgra_.data(), x_special_, y_special_);

    CUDA_ERROR_CHK(hipDeviceSynchronize());

    std::copy(bgra_.begin(), bgra_.end(), pixels);
  } else {
    floats_to_bgras_cpu(width_, height_, colors_.data(), pixels, x_special_, y_special_);
  }
}

template class Renderer<ExecutionModel::CPU>;
template class Renderer<ExecutionModel::GPU>;
} // namespace ray
