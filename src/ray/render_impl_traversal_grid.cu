#include "hip/hip_runtime.h"
#include "ray/projection_impl.h"
#include "ray/render_impl.h"
#include "ray/render_impl_utils.h"
#include "ray/sort_actions.h"

#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/scan.h>
#include <thrust/transform.h>

#include <chrono>
#include <dbg.h>

namespace ray {
using namespace detail;

struct CreateTraversal : public thrust::binary_function<int, int, Traversal> {
  HOST_DEVICE Traversal operator()(int start, int end) {
    return Traversal(start, end);
  }
};

template <ExecutionModel execution_model>
TraversalGridsRef RendererImpl<execution_model>::traversal_grids(
    bool show_times, const Eigen::Projective3f &world_to_film,
    Span<const scene::ShapeData, false> shapes,
    Span<const scene::Light, false> lights) {
  namespace chr = std::chrono;
  const auto setup_traversal_grid = chr::high_resolution_clock::now();

  moved_shapes_.resize(shapes.size());
  std::copy(shapes.begin(), shapes.end(), moved_shapes_.begin());


  const Eigen::Array3<unsigned> num_divisions(8, 8, 8);

  const Eigen::Array3<unsigned> shifted_1_num_divisions(
      num_divisions[1], num_divisions[2], num_divisions[0]);
  const Eigen::Array3<unsigned> shifted_2_num_divisions(
      num_divisions[2], num_divisions[0], num_divisions[1]);

  const auto shifted_1_num_translations =
      (2 * shifted_1_num_divisions + 3).eval();
  const auto shifted_2_num_translations =
      (2 * shifted_2_num_divisions + 3).eval();

  const auto total_translations =
      (shifted_1_num_translations * shifted_2_num_translations).eval();

  const unsigned total_size = total_translations.sum();

  unsigned num_division_light_x = 32;
  unsigned num_division_light_y = 32;

  traversal_data_.resize(lights.size() + total_size);
  traversal_grids_.resize(1 + lights.size() + total_size);
  shape_grids_.resize(traversal_grids_.size() * shapes.size());

  unsigned traversal_grid_index = 0;
  unsigned start_shape_grids = 0;
  unsigned start_count_index = 0;

  traversal_grids_[traversal_grid_index] = TraversalGrid(
      TriangleProjector(world_to_film.matrix()), Eigen::Array2f(-1, -1),
      Eigen::Array2f(1, 1), block_data_.num_blocks_x, block_data_.num_blocks_y,
      start_shape_grids, start_count_index, 1.0f, 1.0f, false, true);

  start_shape_grids += shapes.size();
  start_count_index += block_data_.num_blocks_x * block_data_.num_blocks_y;

  traversal_grid_index++;

  const auto &max_bound = scene_->getMaxBound();
  const auto &min_bound = scene_->getMinBound();

  const auto center = ((min_bound + max_bound) / 2).eval();
  const auto dims = (max_bound - min_bound).eval();

  auto get_plane = [&](bool is_loc, const Eigen::Vector3f &loc_or_dir) {
    const auto dir = is_loc ? (center - loc_or_dir).eval() : loc_or_dir;
    const auto normalized_directions = dir.array() / dims.array();
    unsigned axis;
    float max_axis_v = std::numeric_limits<float>::lowest();
    for (unsigned test_axis = 0; test_axis < 3; test_axis++) {
      float abs_v = std::abs(normalized_directions[test_axis]);
      if (abs_v > max_axis_v) {
        axis = test_axis;
        max_axis_v = abs_v;
      }
    }

    float projection_value = max_bound[axis];

    return Plane(projection_value, axis);
  };

  Eigen::Affine3f bounding_transform = Eigen::Translation3f(center) *
                                       Eigen::Scaling(dims) *
                                       Eigen::Affine3f::Identity();

  BoundingPoints bounding_cube = get_bounding(bounding_transform);

  auto add_projection =
      [&](bool is_loc, const Eigen::Vector3f &loc_or_dir,
          unsigned num_divisions_x, unsigned num_divisions_y, Plane plane,
          const thrust::optional<std::tuple<Eigen::Array2f, Eigen::Array2f>>
              &projection_surface_min_max,
          float min_dist_multiplier, float max_dist_multiplier) {
        if (is_loc && (loc_or_dir.array() <= max_bound.array()).all() &&
            (loc_or_dir.array() >= min_bound.array()).all()) {
          dbg("INTERNAL POINT PROJECTIONS NOT SUPPORTED");
          abort();
        }

        Eigen::Array2f projected_min =
            Eigen::Array2f(std::numeric_limits<float>::max(),
                           std::numeric_limits<float>::max());
        Eigen::Array2f projected_max =
            Eigen::Array2f(std::numeric_limits<float>::lowest(),
                           std::numeric_limits<float>::lowest());

        TriangleProjector projector(DirectionPlane(loc_or_dir, is_loc, plane));

        if (projection_surface_min_max.has_value()) {
          const auto &[p_min, p_max] = *projection_surface_min_max;
          projected_min = p_min;
          projected_max = p_max;
        } else {
          for (const auto &point : bounding_cube) {
            const auto [projected, _] = project_point(point, projector);
            projected_min = projected_min.cwiseMin(projected);
            projected_max = projected_max.cwiseMax(projected);
          }
        }

        traversal_grids_[traversal_grid_index] = TraversalGrid(
            projector, projected_min, projected_max, num_divisions_x,
            num_divisions_y, start_shape_grids, start_count_index,
            min_dist_multiplier, max_dist_multiplier);

        start_shape_grids += shapes.size();
        start_count_index += num_divisions_x * num_divisions_y;
        traversal_grid_index++;
      };

  for (const auto &light : lights) {
    auto add_light_projection = [&](bool is_loc, Eigen::Vector3f loc_or_dir) {
      auto plane = get_plane(is_loc, loc_or_dir);
      add_projection(is_loc, loc_or_dir, num_division_light_x,
                     num_division_light_y, plane, thrust::nullopt, 1.0f, 1.0f);
    };

    light.visit([&](auto &&light_data) {
      using T = std::decay_t<decltype(light_data)>;
      if constexpr (std::is_same<T, scene::DirectionalLight>::value) {
        add_light_projection(false, light_data.direction);
      } else {
        add_light_projection(true, light_data.position);
      }
    });
  }

  std::array<unsigned, 3> traversal_data_starts;

  Eigen::Array3f multipliers = dims.array() / num_divisions.cast<float>();
  std::array<Eigen::Array2f, 3> min_side_bounds;
  std::array<Eigen::Array2f, 3> max_side_bounds;
  std::array<Eigen::Array2<int>, 3> min_side_diffs;
  std::array<Eigen::Array2<int>, 3> max_side_diffs;

  Eigen::Array3f inverse_multipliers = 1.0f / multipliers;

  auto num_divisions_p_1 = (num_divisions + 1).eval();


  for (uint8_t axis : {0, 1, 2}) {
    traversal_data_starts[axis] = traversal_grid_index - 1;
    uint8_t first_axis = (axis + 1) % 3;
    uint8_t second_axis = (axis + 2) % 3;
    float first_multip = multipliers[first_axis];
    float second_multip = multipliers[second_axis];
    int first_divisions = num_divisions_p_1[first_axis];
    int second_divisions = num_divisions_p_1[second_axis];

    Plane plane(max_bound[axis], axis);

    min_side_bounds[axis] =
        (plane.get_not_axis(min_bound) -
         plane.get_not_axis(multipliers) *
             plane.get_not_axis(num_divisions_p_1).cast<float>()) *
        plane.get_not_axis(inverse_multipliers);
    max_side_bounds[axis] =
        (plane.get_not_axis(max_bound) +
         plane.get_not_axis(multipliers) *
             plane.get_not_axis(num_divisions_p_1).cast<float>()) *
        plane.get_not_axis(inverse_multipliers);
    min_side_diffs[axis] = -plane.get_not_axis(num_divisions_p_1).cast<int>();
    max_side_diffs[axis] = plane.get_not_axis(num_divisions_p_1).cast<int>();

    auto min_other_bounds = plane.get_not_axis(min_bound);
    auto max_other_bounds = plane.get_not_axis(max_bound);

    for (int translation_second = -second_divisions;
         translation_second <= second_divisions; translation_second++) {
      for (int translation_first = -first_divisions;
           translation_first <= first_divisions; translation_first++) {
        Eigen::Vector3f dir;
        dir[axis] = max_bound[axis] - min_bound[axis];
        dir[first_axis] = translation_first * first_multip;
        dir[second_axis] = translation_second * second_multip;

        float general_dist = dir.norm();

        auto reduce_mag = [](int v) {
          if (v > 0) {
            return v - 1;
          } else if (v < 0) {
            return v + 1;
          }

          return 0;
        };

        auto increase_mag = [](int v) {
          if (v > 0) {
            return v + 1;
          } else if (v < 0) {
            return v - 1;
          }

          return 1;
        };

        Eigen::Vector3f dir_shortest;
        dir_shortest[axis] = max_bound[axis] - min_bound[axis];
        dir_shortest[first_axis] = reduce_mag(translation_first) * first_multip;
        dir_shortest[second_axis] =
            reduce_mag(translation_second) * second_multip;

        float shortest_dist = dir_shortest.norm();

        Eigen::Vector3f dir_longest;
        dir_longest[axis] = max_bound[axis] - min_bound[axis];
        dir_longest[first_axis] =
            increase_mag(translation_first) * first_multip;
        dir_longest[second_axis] =
            increase_mag(translation_second) * second_multip;

        float longest_dist = dir_longest.norm();

        float min_dist_multiplier = shortest_dist / general_dist;
        float max_dist_multiplier = longest_dist / general_dist;

        auto dir_other_axis = plane.get_not_axis(dir);

        Eigen::Array2f projected_min =
            min_other_bounds.cwiseMin(dir_other_axis + min_other_bounds);
        Eigen::Array2f projected_max =
            max_other_bounds.cwiseMax(dir_other_axis + max_other_bounds);

        add_projection(
            false, dir,
            num_divisions[first_axis] + unsigned(std::abs(translation_first)),
            num_divisions[second_axis] + unsigned(std::abs(translation_second)),
            plane, std::make_tuple(projected_min, projected_max),
            min_dist_multiplier, max_dist_multiplier);
      }
    }
  }

  if (show_times) {
    dbg(chr::duration_cast<chr::duration<double>>(
            chr::high_resolution_clock::now() - setup_traversal_grid)
            .count());
  }

  const auto project_traversal_grid = chr::high_resolution_clock::now();

  shape_bounds_.resize(shapes.size());

  std::transform(shapes.begin(), shapes.end(), shape_bounds_.begin(),
                 [](const scene::ShapeData &shape) {
                   return get_bounding(shape.get_transform());
                 });

  Span<TraversalGrid, false> grid_span(traversal_grids_.data(),
                                       traversal_grids_.size());

  constexpr bool shape_is_outer = false;
  unsigned block_dim_grid = 2;
  unsigned block_dim_shape = 64;

  if constexpr (execution_model == ExecutionModel::GPU) {
    update_shapes<shape_is_outer>(grid_span, to_span(shape_grids_),
                                  to_const_span(shape_bounds_), shapes.size(),
                                  block_dim_grid, block_dim_shape);
  } else {
    update_shapes_cpu(grid_span, to_span(shape_grids_),
                      to_const_span(shape_bounds_), shapes.size());
  }

  if (show_times) {
    dbg(chr::duration_cast<chr::duration<double>>(
            chr::high_resolution_clock::now() - project_traversal_grid)
            .count());
  }

  const auto count_prefix_sum = chr::high_resolution_clock::now();

  action_starts_.resize(start_count_index);
  thrust::fill_n(to_thrust_iter(action_starts_), action_starts_.size(), 0);

  if constexpr (execution_model == ExecutionModel::GPU) {
    update_counts<shape_is_outer>(grid_span, to_const_span(shape_grids_),
                                  to_span(action_starts_), shapes.size(),
                                  block_dim_grid, block_dim_shape);
  } else {
    update_counts_cpu(grid_span, to_const_span(shape_grids_),
                      to_span(action_starts_), shapes.size());
  }

  unsigned last_size = action_starts_[action_starts_.size() - 1];

  thrust::exclusive_scan(to_thrust_iter(action_starts_),
                         to_thrust_iter(action_starts_) + action_starts_.size(),
                         to_thrust_iter(action_starts_));

  unsigned total_num_actions =
      action_starts_[action_starts_.size() - 1] + last_size;

  action_ends_.resize(action_starts_.size());

  thrust::copy(to_thrust_iter(action_starts_),
               to_thrust_iter(action_starts_) + action_starts_.size(),
               to_thrust_iter(action_ends_));

  actions_.resize(total_num_actions);

  if constexpr (execution_model == ExecutionModel::GPU) {
    add_actions<shape_is_outer>(grid_span, to_const_span(shape_grids_),
                                to_span(action_ends_), to_span(actions_),
                                shapes.size(), block_dim_grid, block_dim_shape);
  } else {
    add_actions_cpu(grid_span, to_const_span(shape_grids_),
                    to_span(action_ends_), to_span(actions_), shapes.size());
  }

  if (show_times) {
    dbg(chr::duration_cast<chr::duration<double>>(
            chr::high_resolution_clock::now() - count_prefix_sum)
            .count());
  }

  const auto get_traversal_sort_actions = chr::high_resolution_clock::now();

  traversals_.resize(action_starts_.size());

  auto transform_to_traversal = [&](const auto &execution_type) {
    thrust::transform(execution_type, to_thrust_iter(action_starts_),
                      to_thrust_iter(action_starts_) + action_starts_.size(),
                      to_thrust_iter(action_ends_), to_thrust_iter(traversals_),
                      [] __host__ __device__(int start, int end) {
                        return Traversal(start, end);
                      });
  };

  if constexpr (execution_model == ExecutionModel::GPU) {
    transform_to_traversal(thrust::device);
  } else {
    transform_to_traversal(thrust::host);
  }

  if (show_times) {
    dbg(chr::duration_cast<chr::duration<double>>(
            chr::high_resolution_clock::now() - get_traversal_sort_actions)
            .count());
  }

  const auto copy_into_traversal_grid = chr::high_resolution_clock::now();

  auto transform_to_disable = [&](auto ptr_type) {
    thrust::transform(to_thrust_iter(traversals_),
                      to_thrust_iter(traversals_) +
                          block_data_.num_blocks_x * block_data_.num_blocks_y,
                      ptr_type,
                      [] __host__ __device__(const Traversal &traversal) {
                        return traversal.end - traversal.start == 0;
                      });
  };

  if constexpr (execution_model == ExecutionModel::GPU) {
    transform_to_disable(thrust::device_ptr<uint8_t>(to_ptr(group_disables_)));
  } else {
    transform_to_disable(to_ptr(group_disables_));
  }

  for (unsigned i = 1; i < traversal_grids_.size(); i++) {
    auto &traversal_grid = traversal_grids_[i];

    traversal_data_[i - 1] = traversal_grid.traversalData();
  }

  if (show_times) {
    dbg(chr::duration_cast<chr::duration<double>>(
            chr::high_resolution_clock::now() - copy_into_traversal_grid)
            .count());
  }

  if (traversal_data_.size() != traversal_grid_index - 1) {
    dbg("INVALID SIZE");
    abort();
  }

  return TraversalGridsRef(
      to_const_span(actions_), to_const_span(traversal_data_),
      to_const_span(traversals_), traversal_data_starts, min_bound, max_bound,
      inverse_multipliers, min_side_bounds, max_side_bounds, min_side_diffs,
      max_side_diffs);
}

template class RendererImpl<ExecutionModel::CPU>;
template class RendererImpl<ExecutionModel::GPU>;
} // namespace ray
