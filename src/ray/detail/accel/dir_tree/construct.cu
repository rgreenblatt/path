#include "hip/hip_runtime.h"
#include "lib/cuda/utils.h"
#include "lib/span_convertable_device_vector.h"
#include "lib/span_convertable_vector.h"
#include "ray/detail/accel/dir_tree/dir_tree_generator.h"
#include "lib/printf_dbg.h"

namespace ray {
namespace detail {
namespace accel {
namespace dir_tree {
template <ExecutionModel execution_model>
void DirTreeGenerator<execution_model>::construct() {
  fill_keys();

  Span<Edge> x_edges(sorted_by_x_edges_);
  Span<Edge> y_edges(sorted_by_y_edges_);

  /* auto */ 

  /* auto tranform_start = thrust::make_transform_iterator(sorted_by_x_edges_.begin(), ) */
  /* thrust::inclusive_scan(InputIterator first, InputIterator last,
   * OutputIterator result) */
  /* sorted_by_x_edges_ */
  
  // TODO consider breaking up data to reduce memory access....

  // approach:
  // - compute segmented scan for ends and starts
  //   - some possiblility to optimize around last split...
  //   - some possiblility to optimize segmentation
  // - test all edge choices and reduce to best choice (test both x and y for
  //   now, later test alternating).
  // - using best choice, write out where new division will be and new
  //   start end initial for dimension along which split was done
  // - filter the other edges using a prefix sum etc
  // - filter sorted by z min and z max using a prefix sum etc

  // generalized segmented prefix sum is important...
  // generalized segmented transform is important...
  // look at how thrust does things and see if special casing is important
  // look at using bit fields or uint8_t to store filter condition

  // 1. approach to segmented ___:
  //  - index globally
  //  - operate from there
  //  - fill global index using dynamic kernel launches where needed and
  //   otherwise looping...

  // 2. approach to segmented ___:
  //  - index into thread block
  //  - either entire thread block or start of data per warp
  //    index into warp
  //  - either entire warp or start of data per index
}

template class DirTreeGenerator<ExecutionModel::CPU>;
template class DirTreeGenerator<ExecutionModel::GPU>;
} // namespace dir_tree
} // namespace accel
} // namespace detail
} // namespace ray
