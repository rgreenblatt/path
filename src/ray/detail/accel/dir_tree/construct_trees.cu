#include "hip/hip_runtime.h"
#include "ray/detail/accel/dir_tree/dir_tree_generator.h"

namespace ray {
namespace detail {
namespace accel {
namespace dir_tree {
template <ExecutionModel execution_model>
void DirTreeGenerator<execution_model>::construct_trees() {
  // approach:
  // - compute scan for ends and starts
  // - test all edge choices and reduce to best choice (test both x and y for
  //   now, later test alternating).
  // - using best choice, write out where new division will be
  // - filter the other edges using a prefix sum etc
  // - filter sorted by z min and z max using a prefix sum etc

  // generalized segmented prefix sum is important...
  // generalized segmented transform is important...
  // look at how thrust does things and see if special casing is important
  // look at using bit fields or uint8_t to store filter condition

  // 1. approach to segmented ___:
  //  - index globally
  //  - operate from there
  //  - fill global index using dynamic kernel launches where needed and
  //   otherwise looping...

  // 2. approach to segmented ___:
  //  - index into thread block
  //  - either entire thread block or start of data per warp
  //    index into warp
  //  - either entire warp or start of data per index

}
template class DirTreeGenerator<ExecutionModel::CPU>;
template class DirTreeGenerator<ExecutionModel::GPU>;
} // namespace dir_tree
} // namespace accel
} // namespace detail
} // namespace ray
