#include "hip/hip_runtime.h"
#include "lib/async_for.h"
#include "lib/printf_dbg.h"
#include "ray/detail/accel/dir_tree/dir_tree_generator_impl.h"
#include "ray/detail/accel/dir_tree/group.h"
#include "ray/detail/accel/dir_tree/left_right_counts.h"

#include <thrust/iterator/discard_iterator.h>
#include <thrust/reduce.h>

namespace ray {
namespace detail {
namespace accel {
namespace dir_tree {
HOST_DEVICE inline float cost_heuristic(unsigned num_left, unsigned num_right,
                                        float prop_left) {
  // surface area heuristic (traversal cost and intersection cost terms
  // will be used elsewhere)
  return prop_left * num_left + (1 - prop_left) * num_right;
}

HOST_DEVICE inline float get_prop_left(float value, float min_value,
                                       float max_value) {
  return (value - min_value) / (max_value - min_value);
}

template <ExecutionModel execution_model>
void DirTreeGeneratorImpl<execution_model>::find_best_edges() {
  Span<const unsigned> keys = current_edges_keys_.get();
  Span<const unsigned> groups = current_edges_groups();
  Span<const unsigned> starts_inclusive(starts_inclusive_);
  Span<const float> edge_values = current_edges_->values();
  Span<const uint8_t> edge_is_min = current_edges_->is_mins();
  Span<const unsigned> open_mins_before_group =
      open_mins_before_group_.first.get();
  Span<const unsigned> num_per_group = num_per_group_.first.get();

  thrust::reduce_by_key(
      thrust_data_[0].execution_policy(), keys.begin(),
      keys.begin() + current_edges_keys_->size(),
      thrust::make_transform_iterator(
          thrust::make_counting_iterator(0u),
          [=] __host__ __device__(const unsigned i) {
            auto key = keys[i];
            auto [start, end] = group_start_end(key, groups);

            float first_value_in_region = edge_values[start];
            float last_value_in_region = edge_values[end - 1];
            float this_value = edge_values[i];

            assert(last_value_in_region >= first_value_in_region);

            float prop_left = get_prop_left(this_value, first_value_in_region,
                                            last_value_in_region);

            unsigned start_inclusive = starts_inclusive[i];
            unsigned index_in_group = i - start;
            auto [num_left, num_right, unused] = left_right_counts(
                index_in_group, start_inclusive, open_mins_before_group[key],
                edge_is_min[i], num_per_group[key]);

            float cost = cost_heuristic(num_left, num_right, prop_left);

            return BestEdge(cost, i);
          }),
      thrust::make_discard_iterator(), best_edges_.data(),
      [] __host__ __device__(const unsigned first, const unsigned second) {
        return first == second;
      },
      [] __host__ __device__(const BestEdge &first, const BestEdge &second)
          -> const BestEdge & { return std::min(first, second); });
}

template class DirTreeGeneratorImpl<ExecutionModel::GPU>;
template class DirTreeGeneratorImpl<ExecutionModel::CPU>;
} // namespace dir_tree
} // namespace accel
} // namespace detail
} // namespace ray
