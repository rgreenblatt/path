#include "hip/hip_runtime.h"
#include "ray/detail/impl/fill.h"
#include "ray/detail/render_impl.h"
#include "ray/detail/render_impl_utils.h"

#include <thrust/fill.h>

namespace ray {
namespace detail {

__global__ void initial_world_space_directions_global(
    BlockData block_data, const Eigen::Vector3f world_space_eye,
    const Eigen::Affine3f m_film_to_world,
    Span<Eigen::Vector3f> world_space_directions) {
  initial_world_space_directions_impl(blockIdx.x, threadIdx.x, block_data,
                                      world_space_eye, m_film_to_world,
                                      world_space_directions);
}

// TODO maybe clean this up....
inline void initial_world_space_directions_cpu(
    BlockData block_data, const Eigen::Vector3f &world_space_eye,
    const Eigen::Affine3f &m_film_to_world,
    Span<Eigen::Vector3f> world_space_directions) {
  for (unsigned block_index = 0; block_index < block_data.generalNumBlocks();
       block_index++) {
    for (unsigned thread_index = 0;
         thread_index < block_data.generalBlockSize(); thread_index++) {
      initial_world_space_directions_impl(block_index, thread_index, block_data,
                                          world_space_eye, m_film_to_world,
                                          world_space_directions);
    }
  }
}

template <typename T>
__global__ void fill_data(T *data, unsigned size, T value) {
  unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    data[index] = value;
  }
}

template <ExecutionModel execution_model>
void RendererImpl<execution_model>::fill(
    const scene::Color &initial_multiplier, const scene::Color &initial_color,
    const Eigen::Affine3f &m_film_to_world) {
  const unsigned general_num_blocks = block_data_.generalNumBlocks();
  const unsigned general_block_size = block_data_.generalBlockSize();

  const Eigen::Vector3f world_space_eye = m_film_to_world.translation();

  if constexpr (execution_model == ExecutionModel::GPU) {
    const unsigned fill_block_size = 256;
    const unsigned fill_num_blocks =
        num_blocks(block_data_.totalSize(), fill_block_size);

    fill_data<<<fill_num_blocks, fill_block_size>>>(
        to_ptr(world_space_eyes_), block_data_.totalSize(), world_space_eye);
    fill_data<<<fill_num_blocks, fill_block_size>>>(to_ptr(color_multipliers_),
                                                    block_data_.totalSize(),
                                                    initial_multiplier);
    fill_data<<<fill_num_blocks, fill_block_size>>>(
        to_ptr(colors_), block_data_.totalSize(), initial_color);

    initial_world_space_directions_global<<<general_num_blocks,
                                            general_block_size>>>(
        block_data_, world_space_eye, m_film_to_world,
        to_span(world_space_directions_));

    CUDA_ERROR_CHK(hipDeviceSynchronize());
  } else {
    std::fill(world_space_eyes_.begin(), world_space_eyes_.end(),
              world_space_eye);
    std::fill(color_multipliers_.begin(), color_multipliers_.end(),
              initial_multiplier);
    std::fill(colors_.begin(), colors_.end(), initial_color);

    initial_world_space_directions_cpu(block_data_, world_space_eye,
                                       m_film_to_world,
                                       to_span(world_space_directions_));
  }
}

template class RendererImpl<ExecutionModel::CPU>;
template class RendererImpl<ExecutionModel::GPU>;
} // namespace detail
} // namespace ray
